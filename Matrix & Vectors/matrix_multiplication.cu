
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__ void matrixMultiplication(int *A, int *B, int *C, int N);

void mm(int *A, int *B, int *C, int N);

int main()
{
	hipEvent_t start, end;
	int ROWS = 2;
	int COLS = 2;

	int *hostA = (int *)malloc(sizeof(int) * ROWS * COLS);
	int *hostB = (int *)malloc(sizeof(int) * ROWS * COLS);
	int *hostC = (int *)malloc(sizeof(int) * ROWS * COLS);

	//initialize matrices A and B by random numbers
	srand(time(0));
	int i, j;
	for (i = 0; i < ROWS; i++)
	{
		for (j = 0; j < COLS; j++)
		{
			hostB[i * COLS + j] = rand() % 30;
			hostA[i * COLS + j] = rand() % 20;
		}
	}

	cout << "Matrix A: " << endl;
	for (i = 0; i < ROWS; i++)
	{
		for (j = 0; j < COLS; j++)
		{
			cout << hostA[i * COLS + j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	cout << "Matrix B: " << endl;
	for (i = 0; i < ROWS; i++)
	{
		for (j = 0; j < COLS; j++)
		{
			cout << hostB[i * COLS + j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	int *deviceA, *deviceB, *deviceC;

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);

	hipMalloc(&deviceA, sizeof(int) * ROWS * COLS);
	hipMalloc(&deviceB, sizeof(int) * ROWS * COLS);
	hipMalloc(&deviceC, sizeof(int) * ROWS * COLS);

	hipMemcpy(deviceA, hostA, sizeof(int) * ROWS * COLS, hipMemcpyHostToDevice);

	hipMemcpy(deviceB, hostB, sizeof(int) * ROWS * COLS, hipMemcpyHostToDevice);

	mm(deviceA, deviceB, deviceC, ROWS);

	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
	{
		printf("Cuda failure %s: ", hipGetErrorString(e));
	}

	hipDeviceSynchronize();

	hipMemcpy(hostC, deviceC, ROWS * COLS * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	//now do actual multiplication
	int N = ROWS;
	int *actual = (int *)malloc(sizeof(int) * ROWS * COLS);
	int sum;
	for (int row = 0; row < ROWS; row++)
	{
		for (int col = 0; col < COLS; col++)
		{
			sum = 0;
			for (int n = 0; n < N; n++)
			{
				sum += hostA[row * N + n] * hostB[n * N + col];
			}
			actual[row * N + col] = sum;
		}
	}

	hipEventRecord(end);
	hipEventSynchronize(end);

	float time = 0;
	hipEventElapsedTime(&time, start, end);

	double error = 0;
	for (int k = 0; k < ROWS * COLS; k++)
	{
		cout << k << ")"
			 << "Expected value = " << actual[k] << " Actual value = " << hostC[k] << "\n";
		error += double(abs(actual[k] - hostC[k]));
	}

	error = sqrt(error);
	cout << "Error = " << error << "\n";
	cout << "The time required for it is: " << time << " seconds" << endl;

	delete[] hostA;
	delete[] hostB;
	delete[] hostC;
}

__global__ void matrixMultiplication(int *A, int *B, int *C, int N)
{

	int ROW = blockIdx.y * blockDim.y + threadIdx.y;
	int COL = blockIdx.x * blockDim.x + threadIdx.x;

	int sum = 0;
	if (ROW < N && COL < N)
	{
		for (int i = 0; i < N; i++)
		{
			sum += A[ROW * N + i] * B[i * N + COL];
		}
		__syncthreads();
		C[ROW * N + COL] = sum;
	}
}

void mm(int *A, int *B, int *C, int N)
{
	dim3 threadsPerblock(N, N);
	dim3 blocksPerGrid(1, 1);

	if (N * N > 512)
	{
		threadsPerblock.x = 512;
		threadsPerblock.y = 512;
		blocksPerGrid.x = ceil(double(N) / double(threadsPerblock.x));
		blocksPerGrid.y = ceil(double(N) / double(threadsPerblock.y));
	}

	cout << "Calling mult:"
		 << "\n";

	matrixMultiplication<<<blocksPerGrid, threadsPerblock>>>(A, B, C, N);
}